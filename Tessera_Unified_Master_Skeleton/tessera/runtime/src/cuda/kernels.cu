#include <hip/hip_runtime.h>
extern "C" {

__global__ void gemm_naive_kernel(const float* __restrict__ A, const float* __restrict__ B, float* __restrict__ C,
                                  int M, int K, int N){
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  if(row<M && col<N){
    float acc = 0.f;
    for(int k=0;k<K;k++) acc += A[row*K + k] * B[k*N + col];
    C[row*N + col] = acc;
  }
}

__global__ void bn_infer_lastdim_kernel(const float* __restrict__ X, const float* __restrict__ mean, const float* __restrict__ var,
                                        const float* __restrict__ gamma, const float* __restrict__ beta, float* __restrict__ Y,
                                        int rows, int C, float eps){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  int total = rows*C;
  if(idx < total){
    int c = idx % C;
    float inv_std = rsqrtf(var[c] + eps);
    Y[idx] = gamma[c] * (X[idx] - mean[c]) * inv_std + beta[c];
  }
}

} // extern "C"
